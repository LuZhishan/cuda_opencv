#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#define N 5

__global__ void gpuSquare(float *d_in, float *d_out) 
{
	int tid = threadIdx.x;
    d_out[tid] = d_in[tid]*d_in[tid];
}

int main()
{
    float h_in[N], h_out[N];
    for (size_t i = 0; i < N; i++)
    {
        h_in[i] = i;
    }
    float *d_in, *d_out;
    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, N * sizeof(float));
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    gpuSquare<<<1, N>>>(d_in, d_out);
    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < N; i++)
    {
        std::cout << h_out[i] << std::endl;
    }
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 50000

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("thread id is %d, block id is %d, tid is %d\n", threadIdx.x, blockIdx.x, tid);
    while (tid < N)
    {
        d_c[tid] = d_a[tid] + d_b[tid]; // 因为是多线程同时添加，所以这里相当于一次性添加了一个grid(512*512)的数据
        tid += blockDim.x * gridDim.x;  // 因为现在只有一个维度，所以后续就要在下一个网格分配数据了
    }
}

int main()
{
	int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

	for (int i = 0; i < N; i++)
    {
		h_a[i] = 2 * i * i;
		h_b[i] = i;
	}

	hipMalloc((void**)&d_a, N * sizeof(int));
	hipMalloc((void**)&d_b, N * sizeof(int));
	hipMalloc((void**)&d_c, N * sizeof(int));
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // 这里描述的是一个grid，包含512个block，每个block里有512个线程然后根据上面开辟的内存分配grid的数量
    gpuAdd<<<512, 512>>>(d_a, d_b, d_c); 
	hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); // 等待所有线程执行完成

    int Correct = 1;
	for (int i = 0; i < N; i++) 
    {
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
			Correct = 0;
            break;
		}
	}
    printf("GPU has computed Sum Correctly\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) 
{
	*d_c = *d_a + *d_b;
}

int main()
{
	int h_a = 1, h_b = 2, h_c;  	// 定义变量用于接收GPU的数据
    int *d_a, *d_b, *d_c;	// 定义一个指针
	// 在GPU上开辟内存
	hipMalloc((void**)&d_a, sizeof(int)); 
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));
	// 将数据拷贝至GPU
	hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
	// 调用GPU函数
	gpuAdd <<<1, 1>>> (d_a, d_b, d_c);	
	// 从GPU取回数据
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost); 
	// 释放GPU资源
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	std::cout << h_c << std::endl;
}
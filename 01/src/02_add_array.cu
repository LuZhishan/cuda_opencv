#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#define N 10

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) 
{
	int tid = blockIdx.x;
	if (tid < N)
		d_c[tid] = d_a[tid] + d_b[tid];
}

int main()
{
    int h_a[N], h_b[N], h_c[N]; 
    for (size_t i = 0; i < N; i++)
    {
        h_a[i] = 2*i*i;
        h_b[i] = i ;
	}
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, N * sizeof(int));
	hipMalloc((void**)&d_b, N * sizeof(int));
	hipMalloc((void**)&d_c, N * sizeof(int));
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    // 这里的N是N个块block, 1是每个块内一个线程
	gpuAdd <<<N, 1>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i++)
    {
        std::cout << h_c[i] << std::endl;
    }
    
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}